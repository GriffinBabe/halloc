#include "hip/hip_runtime.h"
/** @file common.cu implementation of common library for Halloc testing */

#include <limits.h>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <unistd.h>

#include <thrust/iterator/counting_iterator.h>
#include <thrust/device_ptr.h>
#include <thrust/functional.h>
#include <thrust/logical.h>
#include <thrust/sort.h>

#include "common.h"

using namespace thrust;

// parsing options
const char *opts_usage_g = 
	"usage: <test-name> <options>\n"
	"\n"
	"supported options are to be added later\n";

void print_usage_and_exit(int exit_code) {
	printf("%s", opts_usage_g);
	exit(exit_code);
}  // print_usage_and_exit

double parse_double(char *str, double a = 0.0, double b = 1.0) {
	double r;
	if(sscanf(str, "%lf", &r) != 1) {
		fprintf(stderr, "%s is not a double value\n", str);
		print_usage_and_exit(-1);
	}
	if(r < a || r > b) {
		fprintf(stderr, "double value %lf is not in range [%lf, %lf]\n", r, a, b);
		print_usage_and_exit(-1);
	}
	return r;
}  // parse_double

int parse_int(char *str, int a = INT_MIN, int b = INT_MAX) {
	int r;
	if(sscanf(str, "%d", &r) != 1) {
		fprintf(stderr, "%s is not an integer value or too big\n", &r);
		print_usage_and_exit(-1);
	}
	if(r < a || r > b) {
		fprintf(stderr, "integer value %d is not in range [%d, %d]\n", r, a, b);
		print_usage_and_exit(-1);
	}
	return r;
}  // parse_int

char *allocator_types[] = {
	"cuda", "halloc", "scatter", "xmalloc"
};

AllocatorType parse_allocator(char *str) {
	int istr;
	for(istr = 0; istr < AllocatorTopNone - 1; istr++)
		if(!strcmp(str, allocator_types[istr]))
			break;
	istr++;
	if(istr == AllocatorTopNone) {
		printf("%s: invalid allocator name\n", str);
		print_usage_and_exit(-1);
	}
	return (AllocatorType)istr;
}  // parse_allocator

void CommonOpts::parse_cmdline(int argc, char **argv) {
	static const char *common_opts_str = ":ha:m:C:B:R:D:n:t:T:s:S:l:f:p:g:";
	int c;
	int period_sh, ndevices;
	cucheck(hipGetDeviceCount(&ndevices));
	bool nthreads_explicit = false, min_alloc_explicit = false, 
		max_alloc_explicit = false;
	while((c = getopt(argc, argv, common_opts_str)) != -1) {
		switch(c) {
			// general options (and errors)
		case 'h':
			print_usage_and_exit(0);
			break;
		case ':':
			fprintf(stderr, "missing argument for option %c\n", optopt);
			print_usage_and_exit(-1);
			break;
		case '?':
			fprintf(stderr, "unknown option -%c\n", optopt);
			print_usage_and_exit(-1);
			break;

			// allocator options
		case 'a':
			allocator = parse_allocator(optarg);
			break;
		case 'm':
			memory = parse_int(optarg, 4096);
			break;
		case 'C':
			halloc_fraction = parse_double(optarg);
			break;
		case 'B':
			busy_fraction = parse_double(optarg);
			break;
		case 'R':
			roomy_fraction = parse_double(optarg);
			break;
		case 'b':
			sb_sz_sh = parse_int(optarg, 20, 26);
			break;

			// test options
		case 'D':
			device = parse_int(optarg, 0, ndevices - 1);
			break;
		case 'n':
			nthreads = parse_int(optarg, 0);
			nthreads_explicit = true;
			break;
		case 't':
			ntries = parse_int(optarg, 1);
			break;
		case 'T':
			bs = parse_int(optarg, 1, 1024);
			break;
		case 's':
			min_alloc_explicit = true;
			alloc_sz = parse_int(optarg, 0);
			if(max_alloc_explicit) { 
				if(max_alloc_sz < alloc_sz) {
					fprintf(stderr, "max allocation size should be >= " 
									"min allocation	size\n");
					print_usage_and_exit(-1);
				}
			} else
				max_alloc_sz = alloc_sz;
			break;
		case 'S':
			max_alloc_explicit = true;
			max_alloc_sz = parse_int(optarg, 0);
			if(min_alloc_explicit) {
				if(max_alloc_sz < alloc_sz) {
					fprintf(stderr, "max allocation size should be >= " 
									"min allocation	size\n");
					print_usage_and_exit(-1);
				}
			} else
				alloc_sz = max_alloc_sz;
			break;
		case 'l':
			nallocs = parse_int(optarg, 1);
			break;
		case 'f':
			alloc_fraction = parse_double(optarg);
			break;
		case 'p':
			period_sh = parse_int(optarg, 0, 31);
			period_mask = period_sh > 0 ? ((1 << period_sh) - 1) : 0;
			break;
		case 'g':
			group_sh = parse_int(optarg, 0, 31);
			break;

		default:
			fprintf(stderr, "this simply should not happen when parsing options\n");
			print_usage_and_exit(-1);
			break;
		}  // switch
	}

	// cap memory to fraction of device memory
	int device;
	cucheck(hipGetDevice(&device));
	hipDeviceProp_t props;
	cucheck(hipGetDeviceProperties(&props, device));
	size_t dev_memory = props.totalGlobalMem;
	memory = min((unsigned long long)memory, 
							 (unsigned long long)(0.75 * dev_memory));

	// cap number of threads for CUDA allocator
	if(allocator == AllocatorCuda && !nthreads_explicit)
		nthreads = min(nthreads, 32 * 1024);
}  // parse_cmdline

double CommonOpts::total_nallocs(void) {
	return (double)nptrs_cont(nthreads) * nallocs * ntries;
}

double CommonOpts::total_sz(void) {
	return alloc_sz * total_nallocs();
}

struct ptr_is_nz {
	void **ptrs;
	CommonOpts opts;
	__host__ __device__ ptr_is_nz(void **ptrs, const CommonOpts &opts) :
		opts(opts), ptrs(ptrs) {}
	__host__ __device__ bool operator()(int i) { 
		if(opts.is_thread_inactive(i)) 
			return true;
		else
			return ptrs[i] != 0;
	}
};  // ptr_is_nz

bool check_nz(void **d_ptrs, uint nptrs, const CommonOpts &opts) {
	return all_of
		(counting_iterator<int>(0), counting_iterator<int>(nptrs),
		 ptr_is_nz(d_ptrs, opts));
}  // check_nz

/** a helper functor to copy to a contiguous location */
struct copy_cont {
	void **d_from;
	CommonOpts opts;
	uint nptrs_cont;
	__host__ __device__ copy_cont(void **d_from, const CommonOpts &opts) 
		: d_from(d_from), opts(opts) {
		this->nptrs_cont = opts.nptrs_cont(opts.nthreads);
	}
	__host__ __device__ void *operator()(int i) {
		uint period = opts.period(), group = opts.group();
		uint it = i % nptrs_cont, ialloc = i / nptrs_cont;
		return d_from[it / group * (period * group) + it % group + 
									ialloc * opts.nthreads];
	}
};  // copy_cont

/** a helper functor to check whether each pointer has enough room */
struct has_enough_room {
	uint64 *d_ptrs;
	size_t alloc_sz;
	int nptrs;
	__host__ __device__ has_enough_room
	(uint64 *d_ptrs, size_t alloc_sz, int	nptrs) 
		: d_ptrs(d_ptrs), alloc_sz(alloc_sz), nptrs(nptrs) {}
	__host__ __device__ bool operator()(int i) {
		if(i == nptrs - 1)
			return true;
		return d_ptrs[i] + alloc_sz <= d_ptrs[i + 1];
	}
};  // has_enough_room

/** a kernel which simply writes thread id at the address specified by each
		pointer in the passed array */
__global__ void write_tid_k(void **d_ptrs, int nptrs) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid >= nptrs)
		return;
	*(int *)d_ptrs[tid] = tid;
}  // write_tid_k

/** a helper functor to check tid written at each address */
struct check_tid {
	void **d_ptrs;
	__host__ __device__ check_tid(void **d_ptrs) : d_ptrs(d_ptrs) {}
	__host__ __device__ bool operator()(int tid) {
		return *(int *)d_ptrs[tid] == tid;
	}
}; 

bool check_alloc(void **d_ptrs, uint nptrs, const CommonOpts &opts) {
	uint alloc_sz = opts.alloc_sz;
	//uint period = opts.period();
	if(!check_nz(d_ptrs, nptrs, opts)) {
		fprintf(stderr, "cannot allocate enough memory\n");
		return false;
	}
	// first copy into a contiguous location
	void **d_ptrs_cont = 0;
	uint group = opts.group();
	int nptrs_cont = opts.nptrs_cont(nptrs / opts.nallocs) * opts.nallocs;
	cucheck(hipMalloc((void **)&d_ptrs_cont, nptrs_cont * sizeof(void *)));
	
	transform
		(counting_iterator<int>(0), counting_iterator<int>(nptrs_cont),
		 device_ptr<void *>(d_ptrs_cont), copy_cont(d_ptrs, opts));
	// sort the pointers
	device_ptr<uint64> dt_ptrs((uint64 *)d_ptrs_cont);
	sort(dt_ptrs, dt_ptrs + nptrs_cont);
	// check whether each pointer has enough room
	if(!all_of(counting_iterator<int>(0), counting_iterator<int>(nptrs_cont), 
						 has_enough_room((uint64 *)d_ptrs_cont, alloc_sz, nptrs_cont))) {
		fprintf(stderr, "allocated pointers do not have enough room\n");
		cucheck(hipFree(d_ptrs_cont));
		return false;
	} 

	// do write-read test to ensure there are no segfaults
	int bs = 128;
	write_tid_k<<<divup(nptrs_cont, bs), bs>>>(d_ptrs_cont, nptrs_cont);
	bool res = all_of(counting_iterator<int>(0), counting_iterator<int>(nptrs_cont), 
								check_tid(d_ptrs_cont));
	cucheck(hipFree(d_ptrs_cont));
	return res;
}  // check_alloc
