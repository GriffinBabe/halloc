#include "hip/hip_runtime.h"
/** @file throughput.cu throughput test for various memory allocators */

#include <common.h>

#include <limits.h>
#include <math.h>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/** measures malloc throughput */
template<class T>
__global__ void throughput_malloc_k
(CommonOpts opts, void **ptrs) {
	int n = opts.nthreads, i = threadIdx.x + blockIdx.x * blockDim.x;
	if(i >= n || i & opts.period_mask)
		return;
	for(int ialloc = 0; ialloc < opts.nallocs; ialloc++) 
		ptrs[i + n * ialloc] = T::malloc(opts.alloc_sz);
}  // throughput_malloc_k

template<class T>
__global__ void throughput_free_k
(CommonOpts opts, void **ptrs) {
	int n = opts.nthreads, i = threadIdx.x + blockIdx.x * blockDim.x;
	if(i >= n || i & opts.period_mask)
		return;
	for(int ialloc = 0; ialloc < opts.nallocs; ialloc++) 
		T::free(ptrs[i + n * ialloc]);
}  // throughput_free_k

template<class T> class ThroughputTest {
	
public:
	void operator()(CommonOpts opts, bool warmup) {
		// allocate memory
		if(warmup) {
			opts.nthreads = min(4 * opts.bs, opts.nthreads);
			opts.ntries = 1;
		}
		if(!warmup)
			printf("throughput test\n");
		int n = opts.nthreads, bs = opts.bs, grid = divup(n, bs);
		int nptrs = n * opts.nallocs;
		size_t ptrs_sz = nptrs * sizeof(void *);
		void **d_ptrs;
		cucheck(hipMalloc((void **)&d_ptrs, ptrs_sz));
		cucheck(hipMemset(d_ptrs, 0, ptrs_sz));

		double t_malloc = 0, t_free = 0;

		// do testing
		for(int itry = 0; itry < opts.ntries; itry++) {
			// allocate
			double t_malloc_start = omp_get_wtime();
			throughput_malloc_k<T> <<<grid, bs>>>(opts, d_ptrs);
			cucheck(hipGetLastError());
			cucheck(hipStreamSynchronize(0));
			double t_malloc_end = omp_get_wtime();
			t_malloc += t_malloc_end - t_malloc_start;
			// check that pointers are correct
			if(!check_nz(d_ptrs, nptrs)) {
				fprintf(stderr, "cannot allocate enough memory\n");
				exit(-1);
			}
			// free
			double t_free_start = omp_get_wtime();
			throughput_free_k<T> <<<grid, bs>>>(opts, d_ptrs);
			cucheck(hipGetLastError());
			cucheck(hipStreamSynchronize(0));
			double t_free_end = omp_get_wtime();
			t_free += t_free_end - t_free_start;
		}  // for(itry)

		// output latency infos
		if(!warmup) {
			double malloc_throughput = opts.total_nallocs() / t_malloc * 1e-6;
			double free_throughput = opts.total_nallocs() / t_free * 1e-6;
			double pair_throughput = opts.total_nallocs() / (t_malloc + t_free) 
				* 1e-6;
			printf("malloc throughput %.2lf Mmallocs/s\n", malloc_throughput);
			printf("free throughput %.2lf Mfrees/s\n", free_throughput);
			printf("pair throughput %.2lf Mpairs/s\n", pair_throughput);
		}  // output latency infos

		// free memory
		cucheck(hipFree(d_ptrs));		
	}  // operator()
 
};  // LatencyTest

int main(int argc, char **argv) {
	CommonOpts opts;
	run_test<ThroughputTest>(argc, argv, opts);
	return 0;
}  // main
